#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <stdlib.h>
#include "graph.h"
#include "book.h"
#include <iostream>

using namespace std;

float	Run_Kernels(TGraph*);
int		InitMemory(TGraph*);
int		InitDeviceSettings(TGraph*);
void	PrintStat(TGraph*);
void	ReleaseCards(TGraph*);
int		Create_Graph(TGraph*, char path[256], int id=0);

int main(int argc, char* argv[])
{
	cout << "Graph traversal on GPU.\n\n";

	vdata size, memgraph;
	TGraph graph;
	char open[256];

	sprintf((char*)open, "C:\\graphs\\input%d.txt", 0);
	//sprintf((char*)open, "\\\\FILE-SERVER\\raid_root\\graphs\\input%d.txt", 0);
	Create_Graph(&graph, open);
	PrintStat(&graph);
	
	StartIteration(&graph);
	printf("%d from %d vertex travelled.\n", graph.result[0], graph.size);
	//ERROR(hipMemcpy(graph.sited, visited, memvisit, hipMemcpyHostToDevice));
	
	printf("Iterations completed in %.3fms\n", Run_Kernels(&graph));

	printf("%d from %d vertex travelled.\n", graph.result[0], graph.size);
	

	ReleaseCards(&graph);
    return 0;
}

float Run_Kernels(TGraph *self)
{
	float timer;
	hipEvent_t start, stop;

	HANDLE_ERROR(
		hipEventCreate(&start)
		);
	HANDLE_ERROR(
		hipEventCreate(&stop)
		);
	HANDLE_ERROR(
		hipEventRecord(start, 0)
		);
	for (int i = 0; i < self->numdevices; i++)
	{
		self->devices[i];
		hipSetDevice(i);
		Iteration<<<BLOCKS, CPUITERATIONS, 0, self->devices[i].stream>>>
			(self->devices[i].devGraph, self->devices[i].devVisited, self->size, self->devices[i].devResult);
	}

	hipDeviceSynchronize();
	HANDLE_ERROR(
		hipEventRecord(stop, 0)
		);
	HANDLE_ERROR(
		hipEventSynchronize(stop)
		);
	HANDLE_ERROR(
		hipEventElapsedTime(&timer, start, stop)
		);

	return timer;
}

void PrintStat(TGraph *self)
{
	printf("\nsize of vdata : %d\n",		sizeof(vdata));
	printf("vertex in graph : %d\n",		self->size);
	printf("arcs in graph   : %d\n\n",		GetArcsCount(self));
	printf("size of graph   : %3.3fMb\n",	(float)self->memory[0].memgraph/1048576);
	printf("size of visited : %3.3fMb\n",	(float)self->memory[0].memvisit/1048576);
	printf("size of result  : %3.3fMb\n",	(float)self->memory[0].memresult/1048576);
	printf("Total allocated : %3.3fMb\n\n",	(float)(self->memory[0].memgraph+self->memory[0].memvisit+self->memory[0].memresult)/1048576);
}

int InitMemory(TGraph *self)
{
	self->memory.resize(self->numdevices+1);
	vdata checkvis = 0, checkres = 0;

	self->memory[0].memgraph  = (GetArcsCount(self)+2*GetVertexCount(self))*sizeof(vdata);
	self->memory[0].memvisit  =  GetVertexCount(self)*sizeof(char);
	self->memory[0].memresult = (GetVertexCount(self)+1+self->numdevices)*sizeof(vdata);
	
	for (int i = 1; i < self->numdevices+1; i++)
	{
		self->memory[i].memgraph  = self->memory[0].memgraph;
		self->memory[i].memvisit  = (GetVertexCount(self)/self->numdevices)*sizeof(char);
		self->memory[i].memresult = GetVertexCount(self)*sizeof(vdata)/self->numdevices+sizeof(vdata);

		checkvis += self->memory[i].memvisit;
		checkres += self->memory[i].memresult;
	}
	self->memory[1].memresult += self->memory[0].memresult-checkres;
	self->memory[1].memvisit  += self->memory[0].memvisit-checkvis;

	return 0;

}

int InitDeviceSettings(TGraph *self)
{
	hipDeviceProp_t prop;

	self->devices.resize(self->numdevices);

	hipSetDevice(0);
	hipSetDeviceFlags(hipDeviceMapHost);
	hipGetDeviceProperties(&prop, 0);
	hipStreamCreate(&(self->devices[0].stream));
	self->devices[0].DeviceID = 0;
	self->devices[0].start    = 0;
	self->devices[0].stop     = self->size/self->numdevices+self->size%self->numdevices;
	ERROR(hipHostAlloc((void **) &(self->devices[0].result),
						self->memory[1].memresult,
						hipHostMallocWriteCombined|hipHostMallocMapped)
						);
	ERROR(hipHostGetDevicePointer(&(self->devices[0].devResult), self->devices[0].result, 0));
	ERROR(hipHostGetDevicePointer(&self->devices[0].devGraph, self->graph, 0));
	
	self->devices[0].name = (char*)malloc(256);
	sprintf(self->devices[0].name, "%s", prop.name);
	
	ERROR(hipMalloc((void **) &(self->devices[0].devVisited), self->memory[1].memvisit));
	printf("%s binded.\n", self->devices[0].name);

	for (int i = 1; i < self->numdevices; i++)
	{
		hipSetDevice(i);
		hipGetDeviceProperties(&prop, i);
		hipSetDeviceFlags(hipDeviceMapHost);
		hipStreamCreate(&(self->devices[i].stream));
		self->devices[i].DeviceID = i;
		self->devices[i].start    = self->devices[i-1].stop+1;
		self->devices[i].stop     = self->devices[i].start+self->size/self->numdevices;
		ERROR(hipHostAlloc((void **) &(self->devices[i].result),
						self->memory[i+1].memresult,
						hipHostMallocWriteCombined|hipHostMallocMapped)
						);
		ERROR(hipHostGetDevicePointer(&(self->devices[i].devResult), self->devices[i].result, 0));
		self->devices[i].name = (char*)malloc(256);
		sprintf(self->devices[i].name, "%s", prop.name);
		ERROR(hipMalloc((void **) &(self->devices[i].devVisited), self->memory[i+1].memvisit));
		ERROR(hipHostGetDevicePointer(&self->devices[i].devGraph, self->graph, 0));
		printf("%s binded.\n", self->devices[i].name);
	}

	return 0;
}

int Create_Graph(TGraph *self, char path[256], int id)
{
	self->id = id;
	hipSetDevice(0);
	printf("Opening %s\n", path);
	file_input(self, path);
	printf("Graph loaded.\n\n");
	
	ERROR(hipGetDeviceCount(&self->numdevices));

	ERROR(hipHostAlloc((void **) &self->result,
						CPUITERATIONS*BLOCKS*self->numdevices+1,
						hipHostMallocWriteCombined|
						hipHostMallocMapped|
						hipHostMallocPortable
						));

	InitMemory(self);

	ERROR(hipHostAlloc((void **) &self->visited, self->memory[0].memvisit, hipHostMallocWriteCombined|hipHostMallocMapped));

	InitDeviceSettings(self);

	return 0;
}

void ReleaseCards(TGraph *self)
{
	hipDeviceProp_t prop;
	for (int i = 0; i < self->numdevices; i++)
	{
		hipSetDevice(i);
		hipGetDeviceProperties(&prop, i);
		ERROR(hipStreamDestroy(self->devices[i].stream));
		ERROR(hipDeviceReset());
		printf("%s released.\n", prop.name);
	}
}