#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "graph.h"
#include <stdio.h>
#include <stdlib.h>
#include "book.h"

vdata numvertex = 0, numarcs = 0;

__device__ vdata expand(vdata *graph)
{
	return 0;
}
__global__ void Iteration(vdata *devGraph, char *devVisited, vdata size, vdata *devResult)
{
	vdata tid = blockIdx.x*blockDim.x + threadIdx.x;
	vdata thread = threadIdx.x;
	vdata block = gridDim.x*blockDim.x;
	vdata offset, vert, numneig, neig, i;
	__shared__ vdata a[CPUITERATIONS][31];

	offset = tid+1;
	i = 0;

	while (1)
	{
		vert = devResult[offset];
		numneig = devGraph[devGraph[vert]];
		for (int j = 1; j <= numneig; j++)
		{
			neig = devGraph[devGraph[vert]+j];
			if (!devVisited[neig] && (a[tid][0] < 50))
			{
				//atomicAdd(&devResult[0], 1);
				a[thread][0]++;
				a[thread][a[thread][0]] = neig;
				//devResult[0]++;
				//devResult[devResult[0]] = neig;
				devVisited[neig]++;
			}
			if (a[thread][0] == 50)
			{
				break;
			}
		}
		offset += block;
		
		if (!devResult[offset]) break;
	}
	__syncthreads();

	
}


int StartIteration(vdata *graph, char *visited, vdata size, vdata* result)
{
	int i = 0, offset = 1, vert, neig, numneig;
	result[0] = 1;
	result[1] = 0;
	visited[0] = 1;
	while (1)
	{
		vert = result[offset];
		numneig = graph[graph[vert]];
		//printf("%d ", vert);
		for (int j = 1; j <= numneig; j++)
		{
			neig = graph[graph[vert]+j];
			if (!visited[neig])
			{
				result[2+i++] = neig;
				visited[neig]++;
			}
		}
		offset++;
		//if (i+2 == offset) break;
		if (!result[offset]) break;
		/*
		if (i > 1000) 
		{
			printf("iteration limit reached.\n");
			break;
		}*/
		//if (i >= CPUITERATIONS*28) break;
	}
	result[0] += i;

	return 0;
}

vdata* stdin_input()
	{
	vdata len, num, offset, temp;
	vdata *graph;

	printf("Reading stdin.\n");
		
	scanf("%d %d", &numvertex, &numarcs);
	HANDLE_ERROR(
		hipHostAlloc((void **)&graph, (2*numvertex+numarcs)*sizeof(vdata), hipHostMallocWriteCombined|hipHostMallocMapped)
	);
	offset = 0;
	for (vdata i = 0; i < numvertex; i++)
	{
		scanf("%d %d", &num, &len);
		graph[i] = numvertex + offset;
		graph[numvertex+offset] = len;
		offset += 1;
		for (vdata j = 1; j <= len; j++)
		{
			scanf("%d", &temp);
			graph[numvertex+offset] = temp;
			offset += 1;
		}		
	}
	printf("Graph loaded.\n\n");
	return graph;
}

vdata* file_input(struct TGraph *self, char *in)
{
	FILE *fp;
	vdata len, num, offset, temp;
	vdata *graph, *devGraph;
	
	fp = fopen(in, "r");		
	fscanf(fp, "%d %d", &self->size, &self->numarcs);
	ERROR(hipHostMalloc((void **)&graph,
						(2*self->size+self->numarcs)*sizeof(vdata),
						hipHostMallocWriteCombined |
						hipHostMallocMapped |
						hipHostMallocPortable
						));
	ERROR(hipHostGetDevicePointer(&self->devGraph, graph, 0));
	offset = 0;
	for (vdata i = 0; i < numvertex; i++)
	{
		fscanf(fp, "%d %d", &num, &len);
		graph[i] = numvertex + offset;
		graph[numvertex+offset] = len;
		offset += 1;
		for (vdata j = 1; j <= len; j++)
		{
			fscanf(fp, "%d", &temp);
			graph[numvertex+offset] = temp;
			offset += 1;
		}		
	}
	
	return graph;
}

vdata GetVertexCount(struct TGraph *self)
{
	return self->size;
}

vdata GetArcsCount(struct TGraph *self)
{
	return self->numarcs;
}

int check(vdata *graph, vdata size)
{
	vdata t = 0;
	for (vdata ti = 0; ti < size; ti++)
	{
		t += graph[graph[ti]];
	}
	if (t == size)
		return 1;
	else
		return 0;
}