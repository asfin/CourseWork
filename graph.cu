#include "graph.h"

using namespace std;

extern int pref;
extern int _RUNTHREADS;
int AddKernels(TGraph *self, TDevices *devices)
{
	if (pref)
	{
		FullIteration(self);
		self->byCPU = self->result[0];
		self->total = reduce_host(self->visited, self->size);
		*self->reduced = self->total;
		self->byGPU = 0;
		self->overhead = self->result[0] - self->total;
	}
	else
	{
		StartIteration(self);
		self->byCPU = self->result[0];

		if (self->byCPU >= NUMTHREADS*MAXBLOCKS)
		{
			devices->AddGraphToQuery(self);
		} else {
			self->byGPU = 0;
			self->total = reduce_host(self->visited, self->size);
			*self->reduced = self->total;
			cout << "Insufficent vertex to run kernels.\n";

		}
	}

	return 0;
}

vdata GetReduce(TGraph *self)
{
	hipSetDevice(self->device);
	hipDeviceSynchronize();
	self->total = *self->reduced;
	self->byGPU = self->total - self->byCPU;
	self->overhead = self->result[0] - self->total;
	printf("Befor : %d from %d vertex travelled.\n", self->byCPU, self->size);
	printf("After : %d from %d vertex travelled.\n", self->total, self->size);
	printf("By GPU: %d\n", self->byGPU);
	printf("Overhead : %d vertex\n\n", self->overhead);


	return self->byGPU;
}

int TDevices::RunKernels()
{
	int i;
		i = lastused++;
		//i = 0;
		lastused %= numdevices;
			
		hipSetDevice(i);
		vdata *devred, *buffer;
		for (int j = 0; j < bounded; j++)
		{
			if (boundedGraphs[j]->memOnGPU)
			{
				ERROR(hipMalloc((void **) &boundedGraphs[j]->devGraph, boundedGraphs[j]->memory.memgraph));
				ERROR(hipMalloc((void **) &boundedGraphs[j]->devResult, boundedGraphs[j]->memory.memresult));
				ERROR(hipMalloc((void **) &boundedGraphs[j]->devVisited, boundedGraphs[j]->memory.memvisit));
				ERROR(hipMemcpyAsync(boundedGraphs[j]->devGraph, boundedGraphs[j]->graph, boundedGraphs[j]->memory.memgraph, hipMemcpyHostToDevice, devices[i].streams[j]));
				ERROR(hipMemcpyAsync(boundedGraphs[j]->devResult, boundedGraphs[j]->result, boundedGraphs[j]->memory.memresult, hipMemcpyHostToDevice, devices[i].streams[j]));
				//ERROR(hipMemsetAsync(boundedGraphs[j]->devVisited, 0, boundedGraphs[j]->memory.memvisit, devices[i].streams[j]));
				//ERROR(hipMemcpyAsync(boundedGraphs[j]->devVisited, boundedGraphs[j]->visited, boundedGraphs[j]->memory.memvisit, hipMemcpyHostToDevice, devices[i].streams[j]));
			} else {
				cout << boundedGraphs[j]->graph << endl;
				ERROR(hipHostGetDevicePointer(&boundedGraphs[j]->devGraph,   boundedGraphs[j]->graph,   0));
				ERROR(hipHostGetDevicePointer(&boundedGraphs[j]->devResult,  boundedGraphs[j]->result,  0));
				ERROR(hipHostGetDevicePointer(&boundedGraphs[j]->devVisited, boundedGraphs[j]->visited, 0));
			}
			ERROR(hipMalloc((void**)&boundedGraphs[j]->flag, sizeof(char)));
			ERROR(hipMalloc((void**)&boundedGraphs[j]->lock, sizeof(int)));
			ERROR(hipMalloc((void**)&boundedGraphs[j]->start, sizeof(vdata)));
			ERROR(hipHostGetDevicePointer(&boundedGraphs[j]->devReduced,   boundedGraphs[j]->reduced,   0));
			boundedGraphs[j]->device = i;
			
		}
		for (int j = 0; j < bounded; j++)
		{
			InitMem<<<1, 1, 0, devices[i].streams[j]>>>(boundedGraphs[j]->start, boundedGraphs[j]->lock, boundedGraphs[j]->devVisited, boundedGraphs[j]->memory.memvisit);
		}
		for (int j = 0; j < bounded; j++)
		{
				Iteration_cc20<<<MAXBLOCKS, NUMTHREADS, 0, devices[i].streams[j]>>>
						(boundedGraphs[j]->devGraph, boundedGraphs[j]->devResult, boundedGraphs[j]->devVisited, boundedGraphs[j]->start, buffer, boundedGraphs[j]->lock, boundedGraphs[j]->flag);
		}
		for (int j = 0; j < bounded; j++)
		{
			reduce_ccAny<<<1, 512, 0, devices[i].streams[j]>>>(boundedGraphs[j]->devVisited, boundedGraphs[j]->size, boundedGraphs[j]->devReduced);
		}
			//*/
		for (int j = 0; j < bounded; j++)
		{
			if (boundedGraphs[j]->memOnGPU)
			{
				//hipMemcpy(self->graph, self->devGraph, self->memory.memgraph, hipMemcpyDeviceToHost);
				hipMemcpyAsync(boundedGraphs[j]->result, boundedGraphs[j]->devResult, boundedGraphs[j]->memory.memresult, hipMemcpyDeviceToHost, devices[i].streams[j]);
				hipMemcpyAsync(boundedGraphs[j]->visited, boundedGraphs[j]->devVisited, boundedGraphs[j]->memory.memvisit, hipMemcpyDeviceToHost, devices[i].streams[j]);
				hipFreeAsync(boundedGraphs[j]->devGraph);
				hipFreeAsync(boundedGraphs[j]->devResult);
				hipFreeAsync(boundedGraphs[j]->devVisited);
			}
			hipFreeAsync(boundedGraphs[j]->flag);
			hipFreeAsync(boundedGraphs[j]->lock);
			hipFreeAsync(boundedGraphs[j]->start);
		}
	bounded = 0;
	return 0;
}

int TDevices::TransferDataToDevice()
{
	return 0;
}

int TDevices::TransferDataFromDevice()
{
	return 0;
}

int TDevices::AddGraphToQuery(TGraph *self)
{
	boundedGraphs[bounded] = self;
	bounded++;
	if (bounded == maxstreams) RunKernels();
	return 0;
}

TDevices::TDevices()
{
	lastused = 0;
	bounded = 0;
	maxstreams = 4;
	ERROR(hipGetDeviceCount(&numdevices));
	devices.resize(numdevices);
	boundedGraphs.resize(16);
	for (int i = 0; i < numdevices; i++)
	{
		hipSetDevice(i);
		devices[i].streams.resize(maxstreams);
		devices[i].DeviceID = i;
		devices[i].laststream = 0;
		hipGetDeviceProperties(&devices[i].prop, i);
		hipSetDeviceFlags(hipDeviceMapHost);
		for (int j = 0; j < maxstreams; j++)
			hipStreamCreate(&devices[i].streams[j]);
		printf("%s binded.\n", devices[i].prop.name);
	}
	printf("\n");
}

TDevices::~TDevices()
{
	printf("\n");
	for (int i = 0; i < numdevices; i++)
	{
		hipSetDevice(i);
		ERROR(hipDeviceReset());
		printf("%s released.\n", devices[i].prop.name);
	}
}

int TDevices::hipFreeAsync(void *devptr)
{
	devPtrs.push_back(devptr);
	return 0;
}

int TDevices::Clean()
{
	for (int i = 0; i < devPtrs.size(); i++)
	{
		hipFree(devPtrs.back());
		devPtrs.pop_back();
	}
	return 0;
}
